#include "hip/hip_runtime.h"
/*
*    This file is part of pNMPC software.
*    Copyright (c) 2020 GIPSA lab [https://github.com/Kartz4code/pNMPC_CODEGEN]
*
*    Main developer - Karthik Murali Madhavan Rathai
*
*    pNMPC - A Code Generation Software Tool For Implementation of Derivative Free
*    Parameterized NMPC Scheme for Embedded Control Systems
*    The software was developed by Karthik Murali Madhavan Rathai under the supervision of
*    Mazen Alamir and Olivier Sename affiliated to Univ. Grenoble Alpes, CNRS, Grenoble INP,
*    GIPSA lab, 38000 Grenoble, France.
*
* 	 pNMPC software is free software: you can redistribute it and/or modify
*    it under the terms of the GNU General Public License as published by
*    the Free Software Foundation, either version 3 of the License, or
*    (at your option) any later version.
*
*	 pNMPC software is distributed in the hope that it will be useful,
*	 but WITHOUT ANY WARRANTY; without even the implied warranty of
*	 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*	 GNU General Public License for more details.
*
*	 You should have received a copy of the GNU General Public License
* 	 along with pNMPC software.  If not, see <http://www.gnu.org/licenses/>.
*
*/

#pragma once
#include "SQPCUDA_BBO.cuh"
#include <stdio.h>

// Objective interface code
__device__
Real_C funcJInterface(Real_C P[NMAX], const ParaStructC* params, Real_C(*funcJ)(Real_C[NMAX], const ParaStructC* params), Real_C(*funcG)(Real_C[NMAX], const ParaStructC* params), Real_C x, int index)
{
	Real_C P_temp = P[index];
	P[index] = x;
	Real_C P_out = funcJ(P, params);
	P[index] = P_temp;
	return P_out;
}

// Constraints interface code
__device__
Real_C funcGInterface(Real_C P[NMAX], const ParaStructC* params, Real_C(*funcJ)(Real_C[NMAX], const ParaStructC* params), Real_C(*funcG)(Real_C[NMAX], const ParaStructC* params), Real_C x, int index)
{
	Real_C P_temp = P[index];
	P[index] = x;
	Real_C P_out = funcG(P, params);
	P[index] = P_temp;
	return P_out;
}

__device__
bool isInSet(const Interval* I1, const Interval* I2, int index)
{
	return (I2->Imin[index] <= I1->Imin[index] && I2->Imax[index] >= I1->Imax[index]);
}

__device__
Real_C fSol(Real_C Pc, Real_C beta, Real_C bF, Real_C aF, Real_C Pmin_bar, Real_C Pmax_bar)
{
	Real_C Pf_sol;
	if (aF != 0)
		Pf_sol = Pc - (beta*bF / (2.0f * aF));
	else if (aF == 0 && bF <= 0)
		Pf_sol = INT_MAX;
	else if (aF == 0 && bF > 0)
		Pf_sol = INT_MIN;
	return min(Pmax_bar, max(Pmin_bar, Pf_sol));
}

__device__
Real_C QuadFunc(Real_C Pfstar, Real_C pc, Real_C aF, Real_C bF, Real_C cF, Real_C beta)
{
	if (aF == 0 && bF == 0)
		return cF;
	else
		return aF*SQUARE((Pfstar - pc) / beta) + bF*((Pfstar - pc) / beta) + cF;
}

__device__
Real_C SQPValue(Real_C P[NMAX], const ParaStructC* params, Real_C(*funcJ)(Real_C[NMAX], const ParaStructC* params), Real_C(*funcG)(Real_C[NMAX], const ParaStructC* params), int index, const Interval* Isearch, Real_C alpha)
{
	// Setting up variables for constraints and objectives
	Real_C Pmin_bar = max(Isearch->Imin[index], P[index] - alpha);
	Real_C Pmax_bar = min(Isearch->Imax[index], P[index] + alpha);
	Real_C Pc = AVG(Pmin_bar, Pmax_bar);
	Real_C beta = MID(Pmin_bar, Pmax_bar);

	// Objectives
	// Objective positve, 0, negative

	Real_C Jneg = funcJInterface(P, params, funcJ, funcG, Pmin_bar, index);
	Real_C J0 = funcJInterface(P, params, funcJ, funcG, Pc, index);
	Real_C Jpos = funcJInterface(P, params, funcJ, funcG, Pmax_bar, index);

	// Objective quadratic function
	Real_C aJ = AVG(Jpos, Jneg) - J0;
	Real_C bJ = MID(Jneg, Jpos);
	Real_C cJ = J0;

	// Optimal value of the approximated Objective quadratic function
	Real_C PJ_solstar = fSol(Pc, beta, bJ, aJ, Pmin_bar, Pmax_bar);
	Real_C QJ_solstar = QuadFunc(PJ_solstar, Pc, aJ, bJ, cJ, beta);


	// New min/max for objective function
	Real_C QJmin = min(Jneg, min(Jpos, QJ_solstar));
	Real_C QJmax = max(Jneg, max(Jpos, QJ_solstar));


	Real_C PJmin, PJmax;
	// PJmin for objective (Minimum)
	if (QJmin == Jneg)
		PJmin = Pmin_bar;
	else if (QJmin == Jpos)
		PJmin = Pmax_bar;
	else if (QJmin == QJ_solstar)
		PJmin = PJ_solstar;

	// PJmax for objective (Maximum)
	if (QJmax == Jneg)
		PJmax = Pmin_bar;
	else if (QJmax == Jpos)
		PJmax = Pmax_bar;
	else if (QJmax == QJ_solstar)
		PJmax = PJ_solstar;

	return PJmin;
}

__device__
SQP_data* SQPSearchSpace(Real_C P[NMAX], const ParaStructC* params, Real_C(*funcJ)(Real_C[NMAX], const ParaStructC* params), Real_C(*funcG)(Real_C[NMAX], const ParaStructC* params), int index, const Interval* Isearch, Real_C alpha)
{

	// Setting up variables for constraints and objectives
	Real_C Pmin_bar = max(Isearch->Imin[index], P[index] - alpha);
	Real_C Pmax_bar = min(Isearch->Imax[index], P[index] + alpha);
	Real_C Pc = AVG(Pmin_bar, Pmax_bar);
	Real_C beta = MID(Pmin_bar, Pmax_bar);

	// Constraints positive, 0, negative

	Real_C Gminus = funcGInterface(P, params, funcJ, funcG, Pmin_bar, index);
	Real_C G0 = funcGInterface(P, params, funcJ, funcG, Pc, index);
	Real_C Gplus = funcGInterface(P, params, funcJ, funcG, Pmax_bar, index);


	// Constraints quadratic function
	Real_C aG = AVG(Gplus, Gminus) - G0;
	Real_C bG = MID(Gminus, Gplus);
	Real_C cG = G0;

	// Optimal value of the approximated Constraints quadratic function
	Real_C PG_solstar = fSol(Pc, beta, bG, aG, Pmin_bar, Pmax_bar);
	Real_C QG_solstar = QuadFunc(PG_solstar, Pc, aG, bG, cG, beta);

	// New min/max for constraint function
	Real_C QGmin = min(Gminus, min(Gplus, QG_solstar));
	Real_C QGmax = max(Gminus, max(Gplus, QG_solstar));

	Real_C PGmin, PGmax;

	// PGmin for constraints (Minimum)
	if (QGmin == Gminus)
		PGmin = Pmin_bar;
	else if (QGmin == Gplus)
		PGmin = Pmax_bar;
	else if (QGmin == QG_solstar)
		PGmin = PG_solstar;

	// PGmax for constraints (Maximum)
	if (QGmax == Gminus)
		PGmax = Pmin_bar;
	else if (QGmax == Gplus)
		PGmax = Pmax_bar;
	else if (QGmax == QG_solstar)
		PGmax = PG_solstar;

	// Search space 
	Real_C PG0plus, PG0minus;
	if (aG == 0 && bG == 0)
	{
		PG0plus = Pc;
		PG0minus = Pc;
	}
	else
	{
		PG0plus = Pc + beta*max((-bG + sqrtf(DISC(aG, bG, cG))) / (2.0f * aG), (-bG - sqrtf(DISC(aG, bG, cG))) / (2.0f * aG));
		PG0minus = Pc + beta*min((-bG + sqrtf(DISC(aG, bG, cG))) / (2.0f * aG), (-bG - sqrtf(DISC(aG, bG, cG))) / (2.0f * aG));
	}

	// SQP data
	SQP_data UI;
	// Zero crossing interval
	Interval Izero;
	Izero.Imin[index] = PG0minus;
	Izero.Imax[index] = PG0plus;

	// Union_Interval

	if (PG0minus < Pmin_bar && Gminus <= 0)
	{
		UI.I_first.Imin[index] = Pmin_bar;
		UI.I_first.Imax[index] = PG0plus;
		UI.nGz = 1;
	}
	else if (PG0minus < Pmin_bar && Gplus <= 0)
	{
		UI.I_first.Imin[index] = PG0plus;
		UI.I_first.Imax[index] = Pmax_bar;
		UI.nGz = 1;
	}
	else if (PG0plus > Pmax_bar && Gminus <= 0)
	{
		UI.I_first.Imin[index] = Pmin_bar;
		UI.I_first.Imax[index] = PG0minus;
		UI.nGz = 1;
	}
	else if (isInSet(&Izero, Isearch, index) == 1 && aG > 0)
	{
		UI.I_first.Imin[index] = PG0minus;
		UI.I_first.Imax[index] = PG0plus;
		UI.nGz = 1;
	}
	else if (isInSet(&Izero, Isearch, index) == 1 && aG < 0)
	{
		UI.I_first.Imin[index] = Pmin_bar;
		UI.I_first.Imax[index] = PG0minus;
		UI.I_second.Imin[index] = PG0plus;
		UI.I_second.Imax[index] = Pmax_bar;
		UI.nGz = 2;
	}
	else
	{
		UI.I_first.Imin[index] = Isearch->Imin[index];
		UI.I_first.Imax[index] = Isearch->Imax[index];
		UI.nGz = 0;
	}

	// Minimum and maximum of objective and solution over interval
	UI.QGmax = QGmax;
	UI.QGmin = QGmin;
	UI.PGmax = PGmax;
	UI.PGmin = PGmin;

	// Return SQP_data object
	return &UI;
}

__device__
char checkCondition(Real_C P[NMAX], const ParaStructC* params, Real_C(*funcJ)(Real_C[NMAX], const ParaStructC* params), Real_C(*funcG)(Real_C[NMAX], const ParaStructC* params), Real_C PCand, int index, char JorG)
{
	char Condition = 0;
	switch (JorG)
	{
	case 0:
        Real_C funcJPCand = funcJInterface(P, params, funcJ, funcG, PCand, index);
        Real_C funcJPIndex = funcJInterface(P, params, funcJ, funcG, P[index], index);
        Real_C funcGPCand = funcGInterface(P, params, funcJ, funcG, PCand, index);
		if ((funcJPCand < funcJPIndex && funcGPCand <= 0) || (funcJPCand <= funcJPIndex && funcGPCand < 0))
			Condition = 1;
        return Condition;
	case 1:
		if (funcGInterface(P, params, funcJ, funcG, PCand, index) < funcGInterface(P, params, funcJ, funcG, P[index], index))
			Condition = 1;
		return Condition;
	case 2:
		if (funcJInterface(P, params, funcJ, funcG, PCand, index) < funcJInterface(P, params, funcJ, funcG, P[index], index) && funcGInterface(P, params, funcJ, funcG, PCand, index) <= 0)
			Condition = 1;
		return Condition;
	default:
		return Condition;
	}

}

__device__
ResVec runSQP(Real_C* P_in, const ParaStructC* params, Real_C(*funcJ)(Real_C* P, const ParaStructC* params), Real_C(*funcG)(Real_C* P, const ParaStructC* params), optimset_SQP* opt, int j)
{
	Real_C PCand, PCand1, PCand2; char JorG; 

	// Print for debug purposes
	SQP_data SD = *SQPSearchSpace(P_in, params, funcJ, funcG, j, &opt->I_init, opt->alpha[j]);
	if (SD.QGmax <= 0)
	{
		PCand = SQPValue(P_in, params, funcJ, funcG, j, &opt->I_init, opt->alpha[j]);
		JorG = 0;
	}
	else if (SD.QGmin > 0)
	{
		PCand = SD.PGmin;
		JorG = 1;
	}
	else if (SD.QGmax > 0 && SD.QGmin < 0)
	{
		switch (SD.nGz)
		{
		case 0:
			PCand = SQPValue(P_in, params, funcJ, funcG, j, &SD.I_first, opt->alpha[j]);
			break;
		case 1:
			PCand = SQPValue(P_in, params, funcJ, funcG, j, &SD.I_first, opt->alpha[j]);
			break;
		case 2:
			PCand1 = SQPValue(P_in, params, funcJ, funcG, j, &SD.I_first, opt->alpha[j]);
			PCand2 = SQPValue(P_in, params, funcJ, funcG, j, &SD.I_second, opt->alpha[j]);
			if ((funcJInterface(P_in, params, funcJ, funcG, PCand1, j) < funcJInterface(P_in, params, funcJ, funcG, PCand2, j)))
				PCand = PCand1;
			else
				PCand = PCand2;
			break;
		default:
			break;
		}

		if (funcGInterface(P_in, params, funcJ, funcG, PCand, j) > 0)
			JorG = 1;
		else
			JorG = 2;
	}
	ResVec result;
	result.JorG = JorG;
	result.PCand = PCand;
	return result;
}

__global__
void SQPCUDA_BBO(Real_C* P_inDevice, ParaStructC* params, const funcPointer funcJ, const funcPointer funcG, optimset_SQP* opt)
{
	const int j = blockIdx.x*blockDim.x + threadIdx.x; 
	Real_C temp[NMAX];
	if (j < NMAX)
	{
		for (uint16_t i = 0; i < opt->Niter; i++)
		{
            //temp[j] = P_inDevice[j];
			for (uint16_t l = 0; l < NMAX; l++)
				temp[l] = P_inDevice[l];

			// Distribution
			ResVec res = runSQP(temp, params, funcJ, funcG, opt, j);
			if (checkCondition(temp, params, funcJ, funcG, res.PCand, j, res.JorG))
			{
				temp[j] = res.PCand;
				opt->alpha[j] = opt->betaPlus*opt->alpha[j];
			}
			else 
				opt->alpha[j] = max(0.00f, opt->betaMinus*opt->alpha[j]);

            // Consensus
			P_inDevice[j] = temp[j];
		}
	}
}